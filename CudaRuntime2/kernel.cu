#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <glad/glad.h>
#include <GLFW/glfw3.h>


#include <stdio.h>
#include "hip/hip_runtime.h"
#include ""
#include <cuda_gl_interop.h>
#include <vector>

#include "Fish.h"
#include "constants.h"
#include "Shader.h"
#include "Parameters.h"

void framebuffer_size_callback(GLFWwindow* window, int width, int height);
void processInput(GLFWwindow* window);





__global__ void calculatePositionKernel(Fish* fishes,int* dev_indexes,int* dev_headsIndex, float dt, 
	float* vertices, Parameters* parameters) {
	
	int i = threadIdx.x + BLOCK_SIZE * blockIdx.x;
	fishes[i].UpdatePositionKernel(fishes, dev_indexes, dev_headsIndex, dt, parameters);
	fishes[i].SetVertexes(vertices + 12 * i);
}


float mouseX = 0;
float mouseY = 0;

// Mouse button callback
void mouse_button_callback(GLFWwindow* window, int button, int action, int mods)
{
	if (button == GLFW_MOUSE_BUTTON_LEFT && action == GLFW_PRESS)
	{
		double xpos, ypos;
		glfwGetCursorPos(window, &xpos, &ypos);
		std::cout << "Mouse clicked at position: (" << xpos << ", " << ypos << ")\n";
		mouseX = xpos;
		mouseY = ypos;
	}
}

Parameters parameters;

float increase_step = 0.1;
void processInput(GLFWwindow* window)
{
	if (glfwGetKey(window, GLFW_KEY_ESCAPE) == GLFW_PRESS)
		glfwSetWindowShouldClose(window, true);
	if (glfwGetKey(window, GLFW_KEY_SPACE) == GLFW_PRESS)
	{
		parameters.stop_simulation = !parameters.stop_simulation;
		cout << "Space pressed" << endl;
	}

	if (glfwGetKey(window, GLFW_KEY_1) == GLFW_PRESS)
		increase_step = 0.1f;
	if (glfwGetKey(window, GLFW_KEY_2) == GLFW_PRESS)
		increase_step = 1.0f;
	if (glfwGetKey(window, GLFW_KEY_3) == GLFW_PRESS)
		increase_step = 10.0f;
	if (glfwGetKey(window, GLFW_KEY_4) == GLFW_PRESS)
		increase_step = 100.0f;
		
	if (glfwGetKey(window, GLFW_KEY_Q) == GLFW_PRESS)
		parameters.avoidWeight -= increase_step;
	if (glfwGetKey(window, GLFW_KEY_W) == GLFW_PRESS)
		parameters.avoidWeight += increase_step;
	if (glfwGetKey(window, GLFW_KEY_A) == GLFW_PRESS)
		parameters.alignWeight -= increase_step;
	if (glfwGetKey(window, GLFW_KEY_S) == GLFW_PRESS)
		parameters.alignWeight += increase_step;
	if (glfwGetKey(window, GLFW_KEY_Z) == GLFW_PRESS)
		parameters.cohesionWeight -= increase_step;
	if (glfwGetKey(window, GLFW_KEY_X) == GLFW_PRESS)
		parameters.cohesionWeight += increase_step;
	cout << "Avoid: " << parameters.avoidWeight << " Align: " 
		<< parameters.alignWeight << " Cohesion: " << parameters.cohesionWeight << endl;
}

		

int main()
{

	

	// glfw: initialize and configure
	// ------------------------------
	glfwInit();
	glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
	glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 3);
	glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);

#ifdef __APPLE__
	glfwWindowHint(GLFW_OPENGL_FORWARD_COMPAT, GL_TRUE);
#endif

	// glfw window creation
	// --------------------
	GLFWwindow* window = glfwCreateWindow(SCR_WIDTH, SCR_HEIGHT, "LearnOpenGL", NULL, NULL);
	if (window == NULL)
	{
		std::cout << "Failed to create GLFW window" << std::endl;
		glfwTerminate();
		return -1;
	}
	glfwMakeContextCurrent(window);
	glfwSetFramebufferSizeCallback(window, framebuffer_size_callback);

	// glad: load all OpenGL function pointers
	// ---------------------------------------
	if (!gladLoadGLLoader((GLADloadproc)glfwGetProcAddress))
	{
		std::cout << "Failed to initialize GLAD" << std::endl;
		return -1;
	}

	Shader ourShader("shader.vs", "shader.fs");


	const int n = NUM_FISH;
	float vertices[n * 12] = { 0 };

	Fish* fishes = new Fish[n];
	for (int i = 0; i < n; i++) {
		int x = rand() % SCR_WIDTH;
		int y = rand() % SCR_HEIGHT;
		fishes[i].SetCordinates((float)x, (float)y);
	}
	fishes[0].id = 99;

	Fish* dev_fishes;
	float* dev_vertices;

	hipError_t cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudasetdevice failed!  do you have a cuda-capable gpu installed?");
		return 1;
	}


	cudaStatus = hipMalloc((void**)&dev_vertices, n * sizeof(float) * 12);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudamalloc failed!");
		goto Error;
	}
	cudaStatus = hipMemcpy(dev_vertices, vertices, n * sizeof(float) * 12, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudamemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_fishes, n * sizeof(Fish));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudamalloc failed!");
		goto Error;
	}
	cudaStatus = hipMemcpy(dev_fishes, fishes, n * sizeof(Fish), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudamemcpy failed!");
		goto Error;
	}

	int indexes[NUM_FISH];
	int headsIndex[NUM_SQUARES];

	int* dev_indexes;
	int* dev_headsIndex;

	cudaStatus = hipMalloc((void**)&dev_indexes, n * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudamalloc failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_indexes, indexes, n * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudamemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_headsIndex, NUM_SQUARES * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudamalloc failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_headsIndex, headsIndex, NUM_SQUARES * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudamemcpy failed!");
		goto Error;
	}

	Parameters* dev_parameters;

	parameters.avoidWeight = AVOID_WEIGHT;
	parameters.alignWeight = ALIGN_WEIGHT;
	parameters.cohesionWeight = COHESION_WEIGHT;
	parameters.stop_simulation = STOP_SIMULATION;
	parameters.speed = SPEED;
	parameters.maxChangeOfDegreePerSecond = MAX_CHANGE_OF_DEGREE_PER_SECOND;
	parameters.alignAngle = ALIGN_ANGLE;
	parameters.cohesionAngle = COHESION_ANGLE;
	parameters.avoidAngle = AVOID_ANGLE;
	parameters.avoidDistance = AVOID_DISTANCE;
	parameters.cohesionDistance = COHESION_DISTANCE;
	parameters.alignDistance = ALIGN_DISTANCE;

	cudaStatus = hipMalloc((void**)&dev_parameters, sizeof(Parameters));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudamalloc failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_parameters, &parameters, sizeof(Parameters), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudamemcpy failed!");
		goto Error;
	}

	

	glfwSetMouseButtonCallback(window, mouse_button_callback);



	unsigned int  VAO;
	glGenVertexArrays(1, &VAO);
	glBindVertexArray(VAO);

	GLuint VBO;
	glGenBuffers(1, &VBO);
	glBindBuffer(GL_ARRAY_BUFFER, VBO);
	glBufferData(GL_ARRAY_BUFFER, sizeof(vertices), nullptr, GL_DYNAMIC_DRAW);
	hipGraphicsResource* cudaVBO;
	hipGraphicsGLRegisterBuffer(&cudaVBO, VBO, cudaGraphicsMapFlagsWriteDiscard);

	glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, 4 * sizeof(float), (void*)0);
	glEnableVertexAttribArray(0);

	glVertexAttribPointer(1, 1, GL_FLOAT, GL_FALSE, 4 * sizeof(float), (void*)(3 * sizeof(float)));
	glEnableVertexAttribArray(1);

	glBindBuffer(GL_ARRAY_BUFFER, 0);
	glBindVertexArray(0);


	


	// render loop
	// -----------
	double lastTime = glfwGetTime();
	_sleep(10);
	while (!glfwWindowShouldClose(window))
	{
		// input
		// -----
		processInput(window);

		// render
		// ------
		glClearColor(0.2f, 0.3f, 0.3f, 1.0f);
		glClear(GL_COLOR_BUFFER_BIT);

		double currentTime = glfwGetTime();

		size_t num_bytes;

		cudaStatus = hipGraphicsMapResources(1, &cudaVBO, 0);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipGraphicsMapResources failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}

		hipGraphicsResourceGetMappedPointer((void**)&dev_vertices, &num_bytes, cudaVBO);

		// making lists for each mesh
		cudaStatus = hipMemcpy(fishes, dev_fishes, n * sizeof(Fish), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}

		
		vector<vector<int>> heads(NUM_SQUARES);
		for (int i = 0; i < NUM_SQUARES; i++)
			heads[i] = {};

		for (int i = 0; i < n; i++) {
			int index = Fish::calculateIndexOfMesh(fishes[i].GetX(), fishes[i].GetY());
			heads[index].push_back(i);
		}

		int index = 0;
		for (int i = 0; i < NUM_SQUARES; i++) {
			headsIndex[i] = index;
			for (int j = 0; j < heads[i].size(); j++) {
				indexes[index++] = heads[i][j];
			}
		}

		cudaStatus = hipMemcpy(dev_indexes, indexes, n * sizeof(int), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "cudamemcpy failed!");
			goto Error;
		}

		cudaStatus = hipMemcpy(dev_headsIndex, headsIndex, NUM_SQUARES * sizeof(int), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "cudamemcpy failed!");
			goto Error;
		}


		calculatePositionKernel << <NUM_FISH/BLOCK_SIZE, BLOCK_SIZE >> > (dev_fishes, 
			dev_indexes, dev_headsIndex, currentTime - lastTime, dev_vertices, dev_parameters);
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "calculatePositionKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}

		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
			goto Error;
		}

		hipGraphicsUnmapResources(1, &cudaVBO, 0);

		ourShader.use();
		glBindVertexArray(VAO);
		glDrawArrays(GL_TRIANGLES, 0, n * 3);

		glfwSwapBuffers(window);
		glfwPollEvents();

		std::cout << 1 / (currentTime - lastTime) << std::endl;

		lastTime = currentTime;
	}


Error:
	// glfw: terminate, clearing all previously allocated GLFW resources.
	// ------------------------------------------------------------------
	glfwTerminate();
	return 0;
}



// glfw: whenever the window size changed (by OS or user resize) this callback function executes
// ---------------------------------------------------------------------------------------------
void framebuffer_size_callback(GLFWwindow* window, int width, int height)
{
	// make sure the viewport matches the new window dimensions; note that width and 
	// height will be significantly larger than specified on retina displays.
	glViewport(0, 0, width, height);
}