#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <glad/glad.h>
#include <GLFW/glfw3.h>


#include <stdio.h>
#include "hip/hip_runtime.h"
#include ""
#include <cuda_gl_interop.h>
#include <vector>

#include "Fish.h"
#include "constants.h"

void framebuffer_size_callback(GLFWwindow* window, int width, int height);
void processInput(GLFWwindow* window);


const char* vertexShaderSource = "#version 330 core\n"
"layout (location = 0) in vec3 aPos;\n"
"layout(location = 1) in float colorId;\n"
"out float vID;\n"
"void main()\n"
"{\n"
"   gl_Position = vec4(aPos.x, aPos.y, aPos.z, 1.0);\n"
"   vID = colorId;\n"
"}\0";

const char* fragmentShaderSource = "#version 330 core\n"
"in float vID;\n"
"out vec4 FragColor;\n"
"void main()\n"
"{\n"
"if(vID == 0.0f){\n"
"   FragColor = vec4(1.0f, 0.0f, 0.2f, 1.0f);}\n"
"else if(vID == 1.0f){\n"
"   FragColor = vec4(0.0f, 1.0f, 0.0f, 1.0f);}\n"
"else if(vID == 2.0f){\n"
"   FragColor = vec4(0.0f, 0.0f, 1.0f, 1.0f);}\n"
"}\n\0";


__global__ void calculatePositionKernel(Fish* fishes,int* dev_indexes,int* dev_headsIndex,
	const int num_squares, float dt, float* vertices, const int n,bool stop_simulation,
	float avoidWeight, float alignWeight, float cohesionWeight) {
	
	int i = threadIdx.x + BLOCK_SIZE * blockIdx.x;
	fishes[i].UpdatePositionKernel(fishes, n,  dev_indexes, dev_headsIndex, num_squares, dt, 0, 0, 
		avoidWeight, alignWeight, cohesionWeight, stop_simulation);
	fishes[i].SetVertexes(vertices + 12 * i);
}



int mouseX = 0;
int mouseY = 0;
// Mouse button callback
void mouse_button_callback(GLFWwindow* window, int button, int action, int mods)
{
	if (button == GLFW_MOUSE_BUTTON_LEFT && action == GLFW_PRESS)
	{
		double xpos, ypos;
		glfwGetCursorPos(window, &xpos, &ypos);
		std::cout << "Mouse clicked at position: (" << xpos << ", " << ypos << ")\n";
		mouseX = xpos;
		mouseY = ypos;
	}
}

float avoidWeight = 4000;
float alignWeight = 50.6;
float cohesionWeight = 0.3;

bool stop_simulation = false;
float increase_step = 0.1;
void processInput(GLFWwindow* window)
{
	if (glfwGetKey(window, GLFW_KEY_ESCAPE) == GLFW_PRESS)
		glfwSetWindowShouldClose(window, true);
	if (glfwGetKey(window, GLFW_KEY_SPACE) == GLFW_PRESS)
	{
		stop_simulation = !stop_simulation;
		cout << "Space pressed" << endl;
	}

	if (glfwGetKey(window, GLFW_KEY_1) == GLFW_PRESS)
		increase_step = 0.1f;
	if (glfwGetKey(window, GLFW_KEY_2) == GLFW_PRESS)
		increase_step = 1.0f;
	if (glfwGetKey(window, GLFW_KEY_3) == GLFW_PRESS)
		increase_step = 10.0f;
	if (glfwGetKey(window, GLFW_KEY_4) == GLFW_PRESS)
		increase_step = 100.0f;
		
	if (glfwGetKey(window, GLFW_KEY_Q) == GLFW_PRESS)
		avoidWeight -= increase_step;
	if (glfwGetKey(window, GLFW_KEY_W) == GLFW_PRESS)
		avoidWeight += increase_step;
	if (glfwGetKey(window, GLFW_KEY_A) == GLFW_PRESS)
		alignWeight -= increase_step;
	if (glfwGetKey(window, GLFW_KEY_S) == GLFW_PRESS)
		alignWeight += increase_step;
	if (glfwGetKey(window, GLFW_KEY_Z) == GLFW_PRESS)
		cohesionWeight -= increase_step;
	if (glfwGetKey(window, GLFW_KEY_X) == GLFW_PRESS)
		cohesionWeight += increase_step;
	cout << "Avoid: " << avoidWeight << " Align: " << alignWeight << " Cohesion: " << cohesionWeight << endl;
}

struct ListNode {
	Fish* fish;
	int next;
};

int main()
{

	

	// glfw: initialize and configure
	// ------------------------------
	glfwInit();
	glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
	glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 3);
	glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);

#ifdef __APPLE__
	glfwWindowHint(GLFW_OPENGL_FORWARD_COMPAT, GL_TRUE);
#endif

	// glfw window creation
	// --------------------
	GLFWwindow* window = glfwCreateWindow(SCR_WIDTH, SCR_HEIGHT, "LearnOpenGL", NULL, NULL);
	if (window == NULL)
	{
		std::cout << "Failed to create GLFW window" << std::endl;
		glfwTerminate();
		return -1;
	}
	glfwMakeContextCurrent(window);
	glfwSetFramebufferSizeCallback(window, framebuffer_size_callback);

	// glad: load all OpenGL function pointers
	// ---------------------------------------
	if (!gladLoadGLLoader((GLADloadproc)glfwGetProcAddress))
	{
		std::cout << "Failed to initialize GLAD" << std::endl;
		return -1;
	}




	// compiling vertex shader
	unsigned int vertexShader;
	vertexShader = glCreateShader(GL_VERTEX_SHADER);
	glShaderSource(vertexShader, 1, &vertexShaderSource, NULL);
	glCompileShader(vertexShader);
	int success;
	char infoLog[512];
	glGetShaderiv(vertexShader, GL_COMPILE_STATUS, &success);
	if (!success)
	{
		glGetShaderInfoLog(vertexShader, 512, NULL, infoLog);
		std::cout << "ERROR::SHADER::VERTEX::COMPILATION_FAILED\n" << infoLog << std::endl;
	}

	unsigned int fragmentShader;
	fragmentShader = glCreateShader(GL_FRAGMENT_SHADER);
	glShaderSource(fragmentShader, 1, &fragmentShaderSource, NULL);
	glCompileShader(fragmentShader);

	glGetShaderiv(vertexShader, GL_COMPILE_STATUS, &success);
	if (!success)
	{
		glGetShaderInfoLog(vertexShader, 512, NULL, infoLog);
		std::cout << "ERROR::SHADER::VERTEX::COMPILATION_FAILED\n" << infoLog << std::endl;
	}

	// shader program
	unsigned int shaderProgram;
	shaderProgram = glCreateProgram();
	glAttachShader(shaderProgram, vertexShader);
	glAttachShader(shaderProgram, fragmentShader);
	glLinkProgram(shaderProgram);

	glGetProgramiv(shaderProgram, GL_LINK_STATUS, &success);
	if (!success) {
		glGetProgramInfoLog(shaderProgram, 512, NULL, infoLog);
	}
	glDeleteShader(vertexShader);
	glDeleteShader(fragmentShader);



	const int n = NUM_FISH;
	float vertices[n * 12] = { 0 };

	Fish* fishes = new Fish[n];
	for (int i = 0; i < n; i++) {
		int x = rand() % SCR_WIDTH;
		int y = rand() % SCR_HEIGHT;
		fishes[i].SetCordinates((float)x, (float)y);
	}
	fishes[0].id = 99;

	Fish* dev_fishes;
	float* dev_vertices;

	hipError_t cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudasetdevice failed!  do you have a cuda-capable gpu installed?");
		return 1;
	}


	cudaStatus = hipMalloc((void**)&dev_vertices, n * sizeof(float) * 12);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudamalloc failed!");
		goto Error;
	}
	cudaStatus = hipMemcpy(dev_vertices, vertices, n * sizeof(float) * 12, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudamemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_fishes, n * sizeof(Fish));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudamalloc failed!");
		goto Error;
	}
	cudaStatus = hipMemcpy(dev_fishes, fishes, n * sizeof(Fish), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudamemcpy failed!");
		goto Error;
	}

	const int num_squares = num_rows * num_cols;
	int indexes[n];
	int headsIndex[num_squares];

	int* dev_indexes;
	int* dev_headsIndex;

	cudaStatus = hipMalloc((void**)&dev_indexes, n * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudamalloc failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_indexes, indexes, n * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudamemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_headsIndex, num_squares * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudamalloc failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_headsIndex, headsIndex, num_squares * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudamemcpy failed!");
		goto Error;
	}

	

	glfwSetMouseButtonCallback(window, mouse_button_callback);



	unsigned int  VAO;
	glGenVertexArrays(1, &VAO);
	glBindVertexArray(VAO);

	GLuint VBO;
	glGenBuffers(1, &VBO);
	glBindBuffer(GL_ARRAY_BUFFER, VBO);
	glBufferData(GL_ARRAY_BUFFER, sizeof(vertices), nullptr, GL_DYNAMIC_DRAW);
	hipGraphicsResource* cudaVBO;
	hipGraphicsGLRegisterBuffer(&cudaVBO, VBO, cudaGraphicsMapFlagsWriteDiscard);

	glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, 4 * sizeof(float), (void*)0);
	glEnableVertexAttribArray(0);

	glVertexAttribPointer(1, 1, GL_FLOAT, GL_FALSE, 4 * sizeof(float), (void*)(3 * sizeof(float)));
	glEnableVertexAttribArray(1);

	glBindBuffer(GL_ARRAY_BUFFER, 0);
	glBindVertexArray(0);


	


	// render loop
	// -----------
	double lastTime = glfwGetTime();
	_sleep(10);
	while (!glfwWindowShouldClose(window))
	{
		// input
		// -----
		processInput(window);

		// render
		// ------
		glClearColor(0.2f, 0.3f, 0.3f, 1.0f);
		glClear(GL_COLOR_BUFFER_BIT);

		double currentTime = glfwGetTime();

		size_t num_bytes;

		cudaStatus = hipGraphicsMapResources(1, &cudaVBO, 0);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipGraphicsMapResources failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}

		hipGraphicsResourceGetMappedPointer((void**)&dev_vertices, &num_bytes, cudaVBO);

		// making lists for each mesh
		cudaStatus = hipMemcpy(fishes, dev_fishes, n * sizeof(Fish), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}

		
		vector<vector<int>> heads(num_squares);
		for (int i = 0; i < num_squares; i++)
			heads[i] = {};

		for (int i = 0; i < n; i++) {
			int index = Fish::calculateIndexOfMesh(fishes[i].GetX(), fishes[i].GetY());
			heads[index].push_back(i);
		}

		int index = 0;
		for (int i = 0; i < num_squares; i++) {
			headsIndex[i] = index;
			for (int j = 0; j < heads[i].size(); j++) {
				indexes[index++] = heads[i][j];
			}
		}

		cudaStatus = hipMemcpy(dev_indexes, indexes, n * sizeof(int), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "cudamemcpy failed!");
			goto Error;
		}

		cudaStatus = hipMemcpy(dev_headsIndex, headsIndex, num_squares * sizeof(int), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "cudamemcpy failed!");
			goto Error;
		}


		calculatePositionKernel << <NUM_FISH/BLOCK_SIZE, BLOCK_SIZE >> > (dev_fishes, 
			dev_indexes, dev_headsIndex,num_squares, currentTime - lastTime, dev_vertices, n, stop_simulation,
			avoidWeight, alignWeight, cohesionWeight);
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "calculatePositionKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}

		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
			goto Error;
		}

		hipGraphicsUnmapResources(1, &cudaVBO, 0);

		glUseProgram(shaderProgram);
		glBindVertexArray(VAO);
		glDrawArrays(GL_TRIANGLES, 0, n * 3);

		glfwSwapBuffers(window);
		glfwPollEvents();

		std::cout << 1 / (currentTime - lastTime) << std::endl;

		lastTime = currentTime;
	}


Error:
	// glfw: terminate, clearing all previously allocated GLFW resources.
	// ------------------------------------------------------------------
	glfwTerminate();
	return 0;
}



// glfw: whenever the window size changed (by OS or user resize) this callback function executes
// ---------------------------------------------------------------------------------------------
void framebuffer_size_callback(GLFWwindow* window, int width, int height)
{
	// make sure the viewport matches the new window dimensions; note that width and 
	// height will be significantly larger than specified on retina displays.
	glViewport(0, 0, width, height);
}