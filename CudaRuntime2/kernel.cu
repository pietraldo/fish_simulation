#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <glad/glad.h>
#include <GLFW/glfw3.h>


#include <stdio.h>
#include "hip/hip_runtime.h"
#include ""


#include "Fish.h"

void framebuffer_size_callback(GLFWwindow* window, int width, int height);
void processInput(GLFWwindow* window);

// settings
const unsigned int SCR_WIDTH = 800;
const unsigned int SCR_HEIGHT = 600;

const char* vertexShaderSource = "#version 330 core\n"
"layout (location = 0) in vec3 aPos;\n"
"layout(location = 1) in float colorId;\n"
"out float vID;\n"
"void main()\n"
"{\n"
"   gl_Position = vec4(aPos.x, aPos.y, aPos.z, 1.0);\n"
"   vID = colorId;\n"
"}\0";

const char* fragmentShaderSource = "#version 330 core\n"
"in float vID;\n"
"out vec4 FragColor;\n"
"void main()\n"
"{\n"
"if(vID == 0.0f){\n"
"   FragColor = vec4(1.0f, 0.0f, 0.2f, 1.0f);}\n"
"else if(vID == 1.0f){\n"
"   FragColor = vec4(0.0f, 1.0f, 0.0f, 1.0f);}\n"
"else if(vID == 2.0f){\n"
"   FragColor = vec4(0.0f, 0.0f, 1.0f, 1.0f);}\n"
"}\n\0";


//__global__ void calculatePositionKernel(Fish* fishes, float dt, float* vertices,const int n) {
//	int i = threadIdx.x;
//	fishes[i].UpdatePositionKernel(fishes, n,dt);
//	fishes[i].SetVertexes(vertices+9*i);
//}


int mouseX = 0;
int mouseY = 0;
// Mouse button callback
void mouse_button_callback(GLFWwindow* window, int button, int action, int mods)
{
	if (button == GLFW_MOUSE_BUTTON_LEFT && action == GLFW_PRESS)
	{
		double xpos, ypos;
		glfwGetCursorPos(window, &xpos, &ypos);
		std::cout << "Mouse clicked at position: (" << xpos << ", " << ypos << ")\n";
		mouseX = xpos;
		mouseY = ypos;
	}
}

int main()
{
	// glfw: initialize and configure
	// ------------------------------
	glfwInit();
	glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
	glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 3);
	glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);

#ifdef __APPLE__
	glfwWindowHint(GLFW_OPENGL_FORWARD_COMPAT, GL_TRUE);
#endif

	// glfw window creation
	// --------------------
	GLFWwindow* window = glfwCreateWindow(SCR_WIDTH, SCR_HEIGHT, "LearnOpenGL", NULL, NULL);
	if (window == NULL)
	{
		std::cout << "Failed to create GLFW window" << std::endl;
		glfwTerminate();
		return -1;
	}
	glfwMakeContextCurrent(window);
	glfwSetFramebufferSizeCallback(window, framebuffer_size_callback);

	// glad: load all OpenGL function pointers
	// ---------------------------------------
	if (!gladLoadGLLoader((GLADloadproc)glfwGetProcAddress))
	{
		std::cout << "Failed to initialize GLAD" << std::endl;
		return -1;
	}




	// compiling vertex shader
	unsigned int vertexShader;
	vertexShader = glCreateShader(GL_VERTEX_SHADER);
	glShaderSource(vertexShader, 1, &vertexShaderSource, NULL);
	glCompileShader(vertexShader);
	int success;
	char infoLog[512];
	glGetShaderiv(vertexShader, GL_COMPILE_STATUS, &success);
	if (!success)
	{
		glGetShaderInfoLog(vertexShader, 512, NULL, infoLog);
		std::cout << "ERROR::SHADER::VERTEX::COMPILATION_FAILED\n" << infoLog << std::endl;
	}

	unsigned int fragmentShader;
	fragmentShader = glCreateShader(GL_FRAGMENT_SHADER);
	glShaderSource(fragmentShader, 1, &fragmentShaderSource, NULL);
	glCompileShader(fragmentShader);

	glGetShaderiv(vertexShader, GL_COMPILE_STATUS, &success);
	if (!success)
	{
		glGetShaderInfoLog(vertexShader, 512, NULL, infoLog);
		std::cout << "ERROR::SHADER::VERTEX::COMPILATION_FAILED\n" << infoLog << std::endl;
	}

	// shader program
	unsigned int shaderProgram;
	shaderProgram = glCreateProgram();
	glAttachShader(shaderProgram, vertexShader);
	glAttachShader(shaderProgram, fragmentShader);
	glLinkProgram(shaderProgram);

	glGetProgramiv(shaderProgram, GL_LINK_STATUS, &success);
	if (!success) {
		glGetProgramInfoLog(shaderProgram, 512, NULL, infoLog);
	}
	glDeleteShader(vertexShader);
	glDeleteShader(fragmentShader);



	const int n = 100;
	float vertices[n * 12] = {0};

	Fish* fishes= new Fish[n];
	for (int i = 0; i < n; i++) {
		int x = rand() % 800;  
		int y = rand() % 600;  
		fishes[i].SetCordinates((float)x,(float)y);
	}


	/*Fish* dev_fishes;
	float* dev_vertices;
	
	hipError_t cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return 1;
	}

	cudaStatus = hipMalloc((void**)&dev_vertices, n * sizeof(float) * 9);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMemcpy(dev_vertices, vertices, n * sizeof(float) * 9, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_fishes, n * sizeof(Fish));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMemcpy(dev_fishes, fishes, n * sizeof(Fish), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}*/

	
	glfwSetMouseButtonCallback(window, mouse_button_callback);
	

	unsigned int VBO, VAO;
	glGenVertexArrays(1, &VAO);
	glGenBuffers(1, &VBO);
	glBindVertexArray(VAO);
	glBindBuffer(GL_ARRAY_BUFFER, VBO);
	glBufferData(GL_ARRAY_BUFFER, sizeof(vertices), nullptr, GL_STATIC_DRAW);

	
	
	glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, 4 * sizeof(float), (void*)0);
	glEnableVertexAttribArray(0);

	glVertexAttribPointer(1, 1, GL_FLOAT, GL_FALSE, 4 * sizeof(float), (void*)(3 * sizeof(float)));
	glEnableVertexAttribArray(1);

	glBindBuffer(GL_ARRAY_BUFFER, 0);
	glBindVertexArray(0);

	// render loop
	// -----------
	double lastTime = glfwGetTime();
	_sleep(10);
	while (!glfwWindowShouldClose(window))
	{
		// input
		// -----
		processInput(window);

		// render
		// ------
		glClearColor(0.2f, 0.3f, 0.3f, 1.0f);
		glClear(GL_COLOR_BUFFER_BIT);

		double currentTime = glfwGetTime();

		/*calculatePositionKernel <<<1, n >>> (dev_fishes,currentTime-lastTime, dev_vertices,n);
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "calculatePositionKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}

		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
			goto Error;
		}

		cudaStatus = hipMemcpy(vertices, dev_vertices, n* sizeof(float)*9, hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}*/

		for (int i = 0; i < n; i++) {
			fishes[i].UpdatePositionKernel(fishes, n, currentTime-lastTime, mouseX, mouseY);
			fishes[i].SetVertexes(vertices + 12 * i);
		}

		//_sleep(10);
		glBindBuffer(GL_ARRAY_BUFFER, VBO);
		glBufferData(GL_ARRAY_BUFFER, sizeof(vertices), vertices, GL_DYNAMIC_DRAW);
		
		glUseProgram(shaderProgram);
		glBindVertexArray(VAO); 
		glDrawArrays(GL_TRIANGLES, 0, n*3);
		
		glfwSwapBuffers(window);
		glfwPollEvents();
		
		//std::cout << 1/(currentTime - lastTime) << std::endl;

		lastTime = currentTime;
	}


//Error:
	// glfw: terminate, clearing all previously allocated GLFW resources.
	// ------------------------------------------------------------------
	glfwTerminate();
	return 0;
}

// process all input: query GLFW whether relevant keys are pressed/released this frame and react accordingly
// ---------------------------------------------------------------------------------------------------------
void processInput(GLFWwindow* window)
{
	if (glfwGetKey(window, GLFW_KEY_ESCAPE) == GLFW_PRESS)
		glfwSetWindowShouldClose(window, true);
}

// glfw: whenever the window size changed (by OS or user resize) this callback function executes
// ---------------------------------------------------------------------------------------------
void framebuffer_size_callback(GLFWwindow* window, int width, int height)
{
	// make sure the viewport matches the new window dimensions; note that width and 
	// height will be significantly larger than specified on retina displays.
	glViewport(0, 0, width, height);
}